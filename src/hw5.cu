#include "hip/hip_runtime.h"
/*
 * Brendan Niebruegge
 * Peter Dirks
 * Homework 5
 * hw5.cu
 * April 26, 2016
 */

#include "../include/hw5.cuh"

__global__ void median_filter(uint8_t *input, const uint32_t width, const uint32_t height, const uint32_t filter_size) {
    // Get x and y location
    uint32_t x           = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y           = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t window_size = filter_size * filter_size;
    uint32_t pivot =  window_size / 2;
    int range = filter_size / 2;

    // error check
    if((x >= width - range) || (y >= height - range) || ((int)x - range) < 0 || ((int)y - range) < 0) {
        return;
    }
    
    // Allocate neighborhood
    uint8_t* neighborhood = new uint8_t[window_size];

    int i = 0; 
    
    // Fill neighborhood accordinly
    for(int current_x = x - range; current_x <= x + range; current_x++) {
        for(int current_y = y - range; current_y <= y + range; current_y++) {
            neighborhood[i] = input[(current_y * width) + current_x];
            ++i;
        }
    }

    // Bubble sort for first half, choosen over insertion sort due to stop once median found
    for(int k = 1; k <= pivot; k++) {
        int min_index = k;

        for(int j = k + 1; j < window_size; j++) {
            if(neighborhood[j] < neighborhood[min_index]) {
                min_index = j;
            }
        }

        // Swap
        uint8_t temp = neighborhood[k];
        neighborhood[k] = neighborhood[min_index];
        neighborhood[min_index] = temp;
    }

    // Update the image
    input[(y * width) + x] = neighborhood[pivot];

    free(neighborhood);
    
}// end median_filter

double hw5_cuda::device_load(uint8_t **host_image, uint32_t width, uint32_t height, uint32_t filter_size, uint8_t** output){

    unsigned int size = width * height * sizeof(uint8_t);

    // Alloc device 
    uint8_t* device_image = NULL;

    *output = (uint8_t *)malloc(size);

    hipMalloc((void**)&device_image, size);
    hipMemcpy(device_image, *host_image, size, hipMemcpyHostToDevice);

    // dim defined in hw5.cuh, we'll start with a 8x8x1
    dim3 dimBlock(DIM_X, DIM_Y, DIM_Z);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // run filter
    median_filter<<<dimGrid, dimBlock, 0>>>(device_image, width, height, filter_size);

    hipMemcpy(*output, device_image, size, hipMemcpyDeviceToHost);



    return sdkGetTimerValue(&timer);

}// end hw5_cuda::device_load

void hw5_cuda::cpu_filter( uint8_t *host_image, uint8_t *cpu_image, uint32_t width, uint32_t height, uint32_t filter_size ){
    uint32_t window_size = filter_size * filter_size;
    int range = (filter_size)/ 2;

    for (int y = 0; y < height; y++){
        for (int x = 0; x < width; x++){
            uint8_t neighborhood[window_size];
            uint32_t neighborhood_index = 0;
            // zero out neighborhood array
            for (uint32_t k = 0; k < window_size; k++){
                neighborhood[k] = 0;
            }

            // populate filter array
            //std::cout << "populate" << std::endl;
            for(int current_x = x - range; current_x <= x + range; current_x++) {
                for (int current_y = y - range; current_y <= y + range; current_y++) {
                    if((x >= width - range) || (y >= height - range) || ((int)x - range) < 0 || ((int)y - range) < 0) {
                        continue;
                    }
                    neighborhood[neighborhood_index] = host_image[(current_y * (int)width) + current_x];
                    neighborhood_index++;
                    //std::cout << "boop" << std::endl;
                }// end inner populate
            }// end outer populate

            //std::cout << "sorting" << std::endl;
            std::sort( neighborhood, neighborhood + window_size );
            //std::cout << "sorting done" << std::endl;
            cpu_image[x + width * y ] = neighborhood[range];
        } // end col for
    }// end row for

}// end cpu_filter

double hw5_cuda::image_filter_error( uint8_t **host_image, uint8_t *gpu_image, uint32_t width, uint32_t height, uint32_t filter_size ){
    char cpu_file[] = "out_cpu.pgm";
    uint32_t window_size = height * width;

    /*
     * create cpu_image
     */
    uint8_t *cpu_image = (uint8_t*)calloc( window_size, sizeof( uint8_t ) );
    if( cpu_image == NULL ){
        return -1;
    }
    cpu_filter( *host_image, cpu_image, width, height, filter_size );
    /*
     * compare pixels for errors
     */
    uint32_t error_count = 0;
    for( uint32_t i = 0; i < window_size; i++ ){
        if( cpu_image[i] != gpu_image[i] ){
            error_count++;
        }
    }
    /*
     * save cpu image file
     */
    if(sdkSavePGM( cpu_file, cpu_image, width, height ) == false){
        return -1;
    }

    //free(cpu_image);

    return (double) (error_count / window_size);
}// end image_filter_error

void hw5_cuda::timerStart(){
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
}
void hw5_cuda::timerStop(){
    sdkStopTimer(&timer);
    sdkDeleteTimer(&timer);
}
double hw5_cuda::getTime(){
    return sdkGetTimerValue(&timer);
}